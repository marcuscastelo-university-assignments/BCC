/*
Faz a soma global dos elementos de um vetor

Exemplifica o uso de atomicADD()

Para compilar: nvcc 00-soma-elems-vet-atomic.cu -o 00-soma-elems-vet-atomic
Para executar: ./00-soma-elems-vet-atomic

OBS: os valores de tamanho do vetor e o conteudo do vetor 
     estao fixos no codigo

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void soma_elementos(int *vetorA,int *soma,int tam)
{
    //Calcula o índice global da thread
    int idx = threadIdx.x+blockIdx.x*blockDim.x; 
    if (idx < tam) 
	{
        //Faz a soma entre elemento do vetor no índice idx e o conteúdo de soma
        atomicAdd(soma,vetorA[idx]); 
    }  
}

int main(int argc,char **argv)
{
    
    //Declara as variáveis para uso no host
    int i,*vetorA,threadsPerBlock,blocksPerGrid,soma=0; 
    
    //Declara os ponteiros para alocação no device
    int *vetorA_d,*soma_d; 
    
    //Define o tamanho do vetor
    int tam=16; //5000;

    //Define a quantidade de threads por bloco
    threadsPerBlock = 256; 

    //Aloca memoria paginada para o vetor no host
    vetorA=(int *)malloc(tam * sizeof(int)); 

    //Aloca o vetor no device
    hipMalloc((void**)&vetorA_d,tam*(sizeof(int))); 
    //Aloca uma variável para armazenar a soma dos elementos do vetor
    hipMalloc((void**)&soma_d,sizeof(int));
    
    //Preenche o vetor no host
    for(i=0;i<tam;i++)
	{
        vetorA[i]=1;
    }
    
    //Define a quantidade de blocos por grade
    blocksPerGrid=(tam+threadsPerBlock-1)/threadsPerBlock; 

    //Copia o conteúdo do vetor para o device
    hipMemcpy(vetorA_d,vetorA,tam*(sizeof(int)), hipMemcpyHostToDevice); 

    //Copia o conteúdo de soma para o device (soma_d = 0)
    hipMemcpy(soma_d,&soma,sizeof(int), hipMemcpyHostToDevice); 

    //Invoca o kernel com blocksPerGrid blocos e threadsPerBlock threads
    soma_elementos <<<blocksPerGrid,threadsPerBlock>>> (vetorA_d, soma_d, tam);

    //Copia o resultado da soma de volta para o host
    hipMemcpy(&soma,soma_d,sizeof(int), hipMemcpyDeviceToHost);

    //Imprime o resultado no host
    printf("%d\n",soma);
    
    //Desaloca o vetor no host
    free(vetorA);  
    
    //Desaloca os vetores no device
    hipFree(vetorA_d);
    hipFree(soma_d);
}
