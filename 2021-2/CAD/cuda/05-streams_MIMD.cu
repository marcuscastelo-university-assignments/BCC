/*
Faz a soma dos elementos de dois vetores em uma stream e em outra stream faz a multiplicacao de um vetor
por um valor escalar

Exemplifica o uso de diferentes streams (1 e 2) para computacoes 
distintas. Usa cudaMallocHost para alocar memoria nao paginavel 
no host e faz copia assincrona com cudaMemcpyAsync. Usa tambem 
o cudaStreamSynchronize para aguardar toda a stream terminar.

O algoritmo calcula na stream 01 a soma de dois vetores e na stream 2
ele faz a multiplicacao de um vetor por um valor escalar.
O algoritmo divide "tam" elementos por "streams_nr" e encontra "threadsPerGrid" e "blocksPerGrid"
Os vetores no device tem o tamanho de threadsPerGrid.
Caso o nr de streams provoque uma divisao com resto, a ultima grade da stream deve acertar o resto.
Funcionou para teste com stream 03 e tam 16 ou 17 e TPB 2 ou 3

Caso o nr de threads por bloco provoque uma divisao com resto, o algoritmo deve funcionar 
com a fitragem do excesso implementada. Funcionou com alguns testes basicos.

Para compilar: nvcc 05-streams_MIMD.cu -o 05-streams_MIMD
Para executar: ./05-streams_MIMD

OBS: os valores de tamanho do vetor e o conteudo do vetor 
     estao fixos no codigo
*/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

//Kernel que faz a soma de vetores
__global__ void soma(int *vetorA, int *vetorB,int *vetorC,int tam, int iter)
{ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < tam) 
    {
        vetorC[idx]=vetorA[idx]+vetorB[idx];
		printf("Soma: Iter=%d, Blc=%d, Thread=%d, Tam=%d, VetC[%d]=%d \n", iter, blockIdx.x, threadIdx.x, tam, idx, vetorC[idx]);
    }
}

//Kernel que faz a multiplicação de um escalar por um vetor
__global__ void mult_escalar(int *vetorA, int escalar,int tam, int iter)
{ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < tam) 
    {
        vetorA[idx]=escalar*vetorA[idx];
		printf("Mult: Iter=%d, Blc=%d, Thread=%d, Tam=%d, vet[%d]=%d \n", iter, blockIdx.x, threadIdx.x, tam, idx, vetorA[idx]);
    }
}

int main(int argc,char **argv)
{
    //Declara as variáveis de índice
    int i,threadsPerBlock; 

    //Inicializa a seed para geração de números pseudo aleatórios
    srand(time(NULL));

    //Declara os vetores no host
    int *vetorA,*vetorB,*vetorC,*vetorD;
    
    int escalar=10; //rand()%10+1;
    
    //Declara os vetores para uso na primeira stream
    int *vetorA_d1,*vetorB_d1,*vetorC_d1;

    //Declara o vetor para uso na segunda stream
    int *vetorD_d2;

    //Declaração das variáveis do tipo cudaStream_t
    hipStream_t stream1,stream2;

    //Criação das streams
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    //Define o tamanho do vetor
    int tam = 17;     //   16; // 2048;
    
    //Define a quantidade de threads por bloco
    threadsPerBlock = 3;  //2; //256; 

	//Define quantas streams usar
	int streams_nr = 3;
	
	//Define o nr de threads por grade (uma grade por vez na stream)
    int threadsPerGrid=tam/streams_nr;  // threadsPerGrid=8 (16/2) //  =1024 (2048/2)
	
    //Define a quantidade de blocos por grade
    int blocksPerGrid=(threadsPerGrid+threadsPerBlock-1)/threadsPerBlock;  // blockPerStream=4 (8/2) //  =4 (1024/256)
	// (8+1)/2 ==> 4 (4,5)
	// (1024+255)/256 ==> 4  (4,9960)
	//
   
    //Aloca memoria nao paginavel para os vetores no host
    hipHostMalloc((void**)&vetorA,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorB,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorC,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorD,tam*(sizeof(int)), hipHostMallocDefault); 

    //Aloca os vetores no device para a stream 1
    hipMalloc((void**)&vetorA_d1,threadsPerGrid*(sizeof(int))); 
    hipMalloc((void**)&vetorB_d1,threadsPerGrid*(sizeof(int))); 
    hipMalloc((void**)&vetorC_d1,threadsPerGrid*(sizeof(int))); 

    //Aloca os vetores no device para a stream 2
    hipMalloc((void**)&vetorD_d2,threadsPerGrid*(sizeof(int))); 
  
    //Preenche os vetores no host
    for(i=0;i<tam;i++)
	{
        vetorA[i]=i;
        vetorB[i]=0; //i;
        vetorD[i]=i; //10;
    }
    
	printf("Host: tam=%d, streams_nr=%d, TPG=%d, BPG=%d, TPB=%d \n", tam, streams_nr, threadsPerGrid, blocksPerGrid, threadsPerBlock);

    for(i=0;i<tam;i+=threadsPerGrid)
	{
		// caso tenha mais streams que o necessario, precisa acertar o tamanho do bloco na ultima stream
		// essa ultima stream pega apenas o restante para processar; nao o vlr cheio de threadsPerGrid
		if((tam-i)< threadsPerGrid)
			threadsPerGrid = tam - i;


        //copia um bloco de tamanho threadsPerGrid do vetor A do host para o device (stream1)
        hipMemcpyAsync(vetorA_d1,vetorA+i,threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream1);
        //copia um bloco de tamanho threadsPerGrid do vetor B do host para o device (stream1)
        hipMemcpyAsync(vetorB_d1,vetorB+i,threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream1);

        //Invoca o kernel soma passando a stream 1 como argumento
        soma <<<blocksPerGrid,threadsPerBlock,0,stream1>>> (vetorA_d1,vetorB_d1,vetorC_d1,threadsPerGrid,i);

        //Copia um bloco de tamanho threadsPerGrid do resultado da stream 1 de volta para o host 
        hipMemcpyAsync(vetorC+i,vetorC_d1,threadsPerGrid*(sizeof(int)),hipMemcpyDeviceToHost,stream1);


        //copia um bloco de tamanho threadsPerGrid do vetor D do host para o device (stream2)
        hipMemcpyAsync(vetorD_d2,vetorD+i,threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream2);
		
        //Invoca o kernel mult_escalar passando a stream 2 como argumento
        mult_escalar <<<blocksPerGrid,threadsPerBlock,0,stream2>>> (vetorD_d2,escalar,threadsPerGrid,i);
		
        //Copia um bloco de tamanho threadsPerGrid do resultado da stream 2 de volta para o host
        hipMemcpyAsync(vetorD+i,vetorD_d2,threadsPerGrid*(sizeof(int)),hipMemcpyDeviceToHost,stream2);

    }
    //Sincroniza as streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    printf("Soma dos vetores:\n");
    //Imprime o resultado da soma de vetores no host
    for(i=0;i<tam;i++)
	{
       printf("%d ",vetorC[i]);
    }
    printf("\n");
    printf("Multiplicação pelo escalar:\n");

    //Imprime o resultado da multiplicação pelo escalar no host
    for(i=0;i<tam;i++)
	{
        printf("%d ",vetorD[i]);
    }
    printf("\n");
    
    //Desaloca os vetores no host
    hipHostFree(vetorA);
    hipHostFree(vetorB);
    hipHostFree(vetorC); 
    hipHostFree(vetorD); 
    
    //Desaloca os vetores da stream 1
    hipFree(vetorA_d1);
    hipFree(vetorB_d1);
    hipFree(vetorC_d1);

    //Desaloca o vetor da stream 2
    hipFree(vetorD_d2);
    
    //Destroi as streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}