/*
Faz a soma dos elementos de dois vetores 

Exemplifica o uso de cudaMallocHost() para alocar memoria paginada no host e
o uso de cudaFreeHost para desalocar()

Para compilar: nvcc 01-soma-vet-pinned.cu -o 01-soma-vet-pinned
Para executar: ./01-soma-vet-pinned

OBS: os valores de tamanho do vetor e o conteudo do vetor 
     estao fixos no codigo

*/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void soma(int *vetorA, int *vetorB,int *vetorC,int tam)
{ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < tam) 
    {
        vetorC[idx]=vetorA[idx]+vetorB[idx];
    }
}

int main(int argc,char **argv)
{
    int i,*vetorA,*vetorB,*vetorC,threadsPerBlock,blocksPerGrid; 
    int *vetorA_d,*vetorB_d,*vetorC_d;

    int tam= 16; // 5000;
    //Define a quantidade de threads por bloco
    threadsPerBlock = 256; 

    //Aloca os vetores no host
    hipHostMalloc((void**)&vetorA,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorB,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorC,tam*(sizeof(int)), hipHostMallocDefault); 

    //Aloca os vetores no device
    hipMalloc((void**)&vetorA_d,tam*(sizeof(int))); 
    hipMalloc((void**)&vetorB_d,tam*(sizeof(int))); 
    hipMalloc((void**)&vetorC_d,tam*(sizeof(int))); 
    
    //Preenche os vetores no host
    for(i=0;i<tam;i++)
	{
        vetorA[i] = i;
        vetorB[i] = 0; //-i;
    }
    
    //Define a quantidade de blocos por grade
    blocksPerGrid=(tam+threadsPerBlock-1)/threadsPerBlock; 

    //Copia o conteúdo dos vetores para o device
    
    hipMemcpy(vetorA_d,vetorA,tam*(sizeof(int)), hipMemcpyHostToDevice); 
    hipMemcpy(vetorB_d,vetorB,tam*(sizeof(int)), hipMemcpyHostToDevice); 

    //Invoca o kernel com blocksPerGrid blocos e threadsPerBlock threads
    
    soma <<<blocksPerGrid,threadsPerBlock>>> (vetorA_d,vetorB_d,vetorC_d,tam); 

    //Copia o resultado da soma de volta para o host
    hipMemcpy(vetorC,vetorC_d,tam*(sizeof(int)), hipMemcpyDeviceToHost);

    //Imprime o resultado no host
    for(i=0;i<tam;i++)
	{
       printf("%d ",vetorC[i]);
    }
	printf("\n");
	
    
    //Desaloca os vetores no host
    hipHostFree(vetorA);
    hipHostFree(vetorB);
    hipHostFree(vetorC); 
    
    //Desaloca os vetores no device
    hipFree(vetorA_d);
    hipFree(vetorB_d);
    hipFree(vetorC_d);
}