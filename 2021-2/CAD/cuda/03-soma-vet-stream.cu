/*
Faz a soma dos elementos de dois vetores 

Exemplifica o uso de diferentes streams com cudaMallocHost
para alocar memoria no host nao paginavel e copia assincrona 
com cudaMemcpyAsync. Usa tambem o cudaStreamSynchronize para
aguardar toda a stream terminar.
O algoritmo divide "tam" elementos por "streams_nr" e encontra "threadsPerGrid" e "blocksPerGrid"
O vetor no device tem o tamanho de threadsPerGrid.
Caso o nr de streams provoque uma divisao com resto, a ultima grade da stream deve acertar o resto.
Funcionou para teste com stream 03 e tam 16 ou 17 e TPB 2 ou 3

Caso o nr de threads por bloco provoque uma divisao com resto, o algoritmo deve funcionar 
com a fitragem do excesso implementada. Funcionou com alguns testes básicos.


Para compilar: nvcc 03-soma-vet-stream.cu -o 03-soma-vet-stream
Para executar: ./03-soma-vet-stream

OBS: os valores de tamanho do vetor e o conteudo do vetor 
     estao fixos no codigo
*/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void soma(int *vetorA, int *vetorB,int *vetorC, int tam, int iter)
{ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < tam) 
    {
        vetorC[idx]=vetorA[idx]+vetorB[idx];
		printf("Iter=%d, Blc=%d, Thread=%d, Tam=%d, VetC[%d]=%d \n", iter, blockIdx.x, threadIdx.x, tam, idx, vetorC[idx]);
    }
}

int main(int argc,char **argv)
{
    int i,*vetorA,*vetorB,*vetorC,threadsPerBlock; 
    int *vetorA_d,*vetorB_d,*vetorC_d;

    //Declaração da variável do tipo cudaStream_t
    hipStream_t stream;

    //Criação da stream
    hipStreamCreate(&stream);

    //Define o tamanho do vetor, multiplo de 256
    int tam = 16;     //   16; // 2048;
    
    //Define a quantidade de threads por bloco
    threadsPerBlock = 2;  //2; //256; 

	//Define quantas streams usar
	int streams_nr = 2;
	
	//Define o nr de threads por grade (uma grade por vez na stream)
    int threadsPerGrid=tam/streams_nr;  // threadsPerGrid=8 (16/2) //  =1024 (2048/2)

	
    //Define a quantidade de blocos por grade
    int blocksPerGrid=(threadsPerGrid+threadsPerBlock-1)/threadsPerBlock;  // blockPerStream=4 (8/2) //  =4 (1024/256)
	// (8+1)/2 ==> 4 (4,5)
	// (1024+255)/256 ==> 4  (4,9960)
	//

    //Aloca memoria nao paginavel para os vetores no host
    hipHostMalloc((void**)&vetorA,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorB,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorC,tam*(sizeof(int)), hipHostMallocDefault); 

    //Aloca os vetores no device
    hipMalloc((void**)&vetorA_d, threadsPerGrid*(sizeof(int))); 
    hipMalloc((void**)&vetorB_d, threadsPerGrid*(sizeof(int))); 
    hipMalloc((void**)&vetorC_d, threadsPerGrid*(sizeof(int))); 
    
    //Preenche os vetores no host
    for(i=0;i<tam;i++)
	{
        vetorA[i]=i;
        vetorB[i]=0;
    }
    
	printf("Host: tam=%d, streams_nr=%d, TPG=%d, BPG=%d, TPB=%d \n", tam, streams_nr, threadsPerGrid, blocksPerGrid, threadsPerBlock);

	// a cada iteracao desloca o bloco em threadsPerGrid itens
	// equivale a deslocar streams_nr
    for(i=0; i<tam; i+=threadsPerGrid)    //i+=8  //i+1024
	{
		// caso tenha mais streams que o necessario, precisa acertar o tamanho do bloco na ultima stream
		// essa ultima stream pega apenas o restante para processas; nao o vlr cheios de threadsPerGrid
		if((tam-i)< threadsPerGrid)
			threadsPerGrid = tam - i;
			
        //copia um bloco de tamanho threadsPerGrid do vetor A do host para o device
        hipMemcpyAsync(vetorA_d,vetorA+i, threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream);
		
        //copia um bloco de tamanho threadsPerGrid do vetor B do host para o device
        hipMemcpyAsync(vetorB_d,vetorB+i, threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream);
		
        //Invoca o kernel soma passando a stream como argumento
        soma <<<blocksPerGrid,threadsPerBlock,0,stream>>> (vetorA_d,vetorB_d,vetorC_d,threadsPerGrid,i);
		// <<<4,2>> para alguns dos exemplos acima.
		
        //Copia um bloco de tamanho threadsPerGrid do resultado de volta para o host
        hipMemcpyAsync(vetorC+i,vetorC_d,threadsPerGrid*(sizeof(int)),hipMemcpyDeviceToHost,stream);
		
    }
    //Sincroniza a stream
    hipStreamSynchronize(stream);

    //Imprime o resultado no host
    for(i=0;i<tam;i++)
	{
       printf("%d ",vetorC[i]);
    }
    
    //Desaloca os vetores no host
    hipHostFree(vetorA);
    hipHostFree(vetorB);
    hipHostFree(vetorC); 
    
    //Desaloca os vetores no device
    hipFree(vetorA_d);
    hipFree(vetorB_d);
    hipFree(vetorC_d);
    
    //Destroi a stream
    hipStreamDestroy(stream);
}