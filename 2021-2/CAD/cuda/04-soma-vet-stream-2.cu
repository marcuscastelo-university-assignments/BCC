/*
Faz a soma dos elementos de dois vetores 

Exemplifica o uso de diferentes streams (1 e 2) com cudaMallocHost
para alocar memoria no host nao paginavel e copia assincrona 
com cudaMemcpyAsync. Usa tambem o cudaStreamSynchronize para
aguardar toda a stream terminar.

O algoritmo divide "tam" elementos por "streams_nr*2" e encontra "threadsPerGrid" e "blocksPerGrid".
Sao invocadas duas streams (1 e 2) a cada iteracao do loop for, por isso a multiplicacao por 2.
O vetor no device tem o tamanho de threadsPerGrid.
Caso o nr de streams provoque uma divisao com resto, a ultima grade da stream acerta o resto
Caso o nr de threads por bloco provoque uma divisao com resto, o algoritmo funciona fitra o excesso


Para compilar: nvcc 04-soma-vet-stream-2.cu -o 04-soma-vet-stream-2
Para executar: ./04-soma-vet-stream-2

OBS: os valores de tamanho do vetor e o conteudo do vetor 
     estao fixos no codigo
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void soma(int *vetorA, int *vetorB, int *vetorC, int tam, int iter, int strm)
{ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < tam) 
    {
        vetorC[idx]=vetorA[idx]+vetorB[idx];
		printf("Iter=%d, Stream=%d, Blc=%d, Thread=%d, Tam=%d, VetC[%d]=%d \n", iter, strm, blockIdx.x, threadIdx.x, tam, idx, vetorC[idx]);

    }
}

int main(int argc,char **argv)
{
    int i,*vetorA,*vetorB,*vetorC,threadsPerBlock; 
    
    //Declara os vetores para uso na primeira stream
    int *vetorA_d1,*vetorB_d1,*vetorC_d1;

    //Declara os vetores para uso na segundo stream
    int *vetorA_d2,*vetorB_d2,*vetorC_d2;

    //Declaração da variável do tipo cudaStream_t
    hipStream_t stream1,stream2;

    //Criação das streams
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    //Define o tamanho do vetor, multiplo de 256
    int tam = 16;     //   16; // 2048;
    
    //Define a quantidade de threads por bloco
    threadsPerBlock = 2;  //2; //256; 

	//Define quantas streams usar
	int streams_nr = 2;
	
	//Define o nr de threads por grade (uma grade por vez na stream)
	// divide por 2 pq sao executadas duas grades por iteracao do for abaixo.
	// cada grade ira executar em uma stream diferente (stream 1 e stream 2)
    int threadsPerGrid=(tam/streams_nr)/2;  // threadsPerGrid=4 (16/2)/2 //  =512 (2048/2)/2
	// 17 / 2 / 2 = 4 (8,5 / 2 = 4,25)
	
    //Define a quantidade de blocos por grade
    int blocksPerGrid=(threadsPerGrid+threadsPerBlock-1)/threadsPerBlock;  // blockPerStream=4 (8/2) //  =4 (1024/256)
	// (4+1)/2 ==> 2 (2,5)
	// (512+255)/256 ==> 2  (2,9960)
	// (4+1)/2 ==> 2  (2,5)


    //Aloca os vetores no host usando pinned pages 
    hipHostMalloc((void**)&vetorA,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorB,tam*(sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&vetorC,tam*(sizeof(int)), hipHostMallocDefault); 

    //Aloca os vetores no device para a stream 1
    hipMalloc((void**)&vetorA_d1,threadsPerGrid*(sizeof(int))); 
    hipMalloc((void**)&vetorB_d1,threadsPerGrid*(sizeof(int))); 
    hipMalloc((void**)&vetorC_d1,threadsPerGrid*(sizeof(int))); 

    //Aloca os vetores no device para a stream 2
    hipMalloc((void**)&vetorA_d2,threadsPerGrid*(sizeof(int))); 
    hipMalloc((void**)&vetorB_d2,threadsPerGrid*(sizeof(int))); 
    hipMalloc((void**)&vetorC_d2,threadsPerGrid*(sizeof(int))); 
    
    //Preenche os vetores no host
    for(i=0;i<tam;i++)
	{
        vetorA[i]= i;
        vetorB[i]= 0; //-i;
    }

	printf("Host: tam=%d, streams_nr=%d, TPG=%d, BPG=%d, TPB=%d \n", tam, streams_nr, threadsPerGrid, blocksPerGrid, threadsPerBlock);

    
    for(i=0;i<tam;i+=threadsPerGrid*2)   // i+=4*2   // i+=512*2   (*2 pq sao duas streams por iter)
	{
		// caso tenha mais streams que o necessario, precisa acertar o tamanho do bloco na ultima stream
		// essa ultima stream pega apenas o restante para processas; nao o vlr cheios de threadsPerGrid
		if((tam-i)< (threadsPerGrid*2))
			threadsPerGrid = tam - i;

        //copia um bloco de tamanho threadsPerGrid do vetor A do host para o device (stream1)
        hipMemcpyAsync(vetorA_d1,vetorA+i,threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream1);
 
		//copia um bloco de tamanho threadsPerGrid do vetor B do host para o device (stream1)
        hipMemcpyAsync(vetorB_d1,vetorB+i,threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream1);

        //Invoca o kernel soma passando a stream 1 como argumento
        soma <<<blocksPerGrid,threadsPerBlock,0,stream1>>> (vetorA_d1,vetorB_d1,vetorC_d1,threadsPerGrid,i,1);

        //Copia um bloco de tamanho bloco do resultado da stream 1 de volta para o host 
        hipMemcpyAsync(vetorC+i,vetorC_d1,threadsPerGrid*(sizeof(int)),hipMemcpyDeviceToHost,stream1);

		if(i+threadsPerGrid < tam)
		{ // se a segunda stream ainda tem computacao a fazer...
			//copia um bloco de tamanho bloco do vetor A do host para o device (stream2)
			hipMemcpyAsync(vetorA_d2,vetorA+i+threadsPerGrid,threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream2);
			//copia um bloco de tamanho bloco do vetor B do host para o device (stream2)
			hipMemcpyAsync(vetorB_d2,vetorB+i+threadsPerGrid,threadsPerGrid*(sizeof(int)),hipMemcpyHostToDevice,stream2);
			
			//Invoca o kernel soma passando a stream 2 como argumento
			soma <<<blocksPerGrid,threadsPerBlock,0,stream2>>> (vetorA_d2,vetorB_d2,vetorC_d2,threadsPerGrid,i,2);
			
			//Copia um bloco de tamanho bloco do resultado da stream 2 de volta para o host
			hipMemcpyAsync(vetorC+i+threadsPerGrid,vetorC_d2,threadsPerGrid*(sizeof(int)),hipMemcpyDeviceToHost,stream2);
		}
    }
    //Sincroniza as streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    //Imprime o resultado no host
    for(i=0;i<tam;i++)
	{
       printf("%d ",vetorC[i]);
    }
    
    //Desaloca os vetores no host
    hipHostFree(vetorA);
    hipHostFree(vetorB);
    hipHostFree(vetorC); 
    
    //Desaloca os vetores da stream 1
    hipFree(vetorA_d1);
    hipFree(vetorB_d1);
    hipFree(vetorC_d1);

    //Desaloca os vetores da stream 2
    hipFree(vetorA_d2);
    hipFree(vetorB_d2);
    hipFree(vetorC_d2);
    
    //Destroi as streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}
