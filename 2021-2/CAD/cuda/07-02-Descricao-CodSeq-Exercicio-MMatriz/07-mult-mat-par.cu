#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG_ACTIVE 0
#define debugCode(code)   \
    do                    \
    {                     \
        if (DEBUG_ACTIVE) \
        {                 \
            code;         \
        }                 \
    } while (0)

#define gpuErrchk(ans)                        \
    do                                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    } while (0)

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void multiplica_matriz(double *devA, double *devB, double *devC, int tam)
{
    extern __shared__ double parcialMults[];

    int i, j, k;
    i = blockIdx.x;
    j = blockIdx.y;
    k = threadIdx.x;

    parcialMults[k] = devA[i * tam + k] * devB[k * tam + j];
    __syncthreads();

    if (k == 0)
    {
        for (int s = 0; s < tam; s++)
            devC[i * tam + j] += parcialMults[s];
    }
}

int main(int argc, char **argv)
{
    // Declara as matrizes
    double *hostA, *hostB, *hostC;
    double *devA, *devB, *devC;
    // Declara as variáveis de tamanho e índice
    int tam, i, j;

    // Lê a dimensão da matriz
    fscanf(stdin, "%d\n", &tam);

    // Aloca as matrizes no host
    hostA = (double *)calloc(tam * tam, sizeof(double));
    hostB = (double *)calloc(tam * tam, sizeof(double));
    hostC = (double *)calloc(tam * tam, sizeof(double));

    // Aloca as matrizes no device
    gpuErrchk(hipMalloc((void **)&devA, tam * tam * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&devB, tam * tam * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&devC, tam * tam * sizeof(double)));

    // Lê as matrizes A e B do stdin
    for (i = 0; i < tam; i++)
        for (j = 0; j < tam; j++)
            fscanf(stdin, "%lf ", &hostA[i * tam + j]);
    for (i = 0; i < tam; i++)
        for (j = 0; j < tam; j++)
            fscanf(stdin, "%lf ", &hostB[i * tam + j]);

    debugCode({
        printf("Matriz A:\n");
        for (i = 0; i < tam; i++)
        {
            for (j = 0; j < tam; j++)
                printf("%.1lf ", hostA[i * tam + j]);
            printf("\n");
        }

        printf("Matriz B:\n");
        for (i = 0; i < tam; i++)
        {
            for (j = 0; j < tam; j++)
                printf("%.1lf ", hostB[i * tam + j]);
            printf("\n");
        }
    });

    // Copia as matrizes para o device
    gpuErrchk(hipMemcpy(devA, hostA, tam * tam * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devB, hostB, tam * tam * sizeof(double), hipMemcpyHostToDevice));

    dim3 dimBlock(tam, tam);
    dim3 tpb(tam);

    // Executa o código CUDA (kernel multiplica_matriz)
    multiplica_matriz<<<dimBlock, tpb, tam>>>(devA, devB, devC, tam);

    // --------------------------------------------------
    // Código antigo (sequencial)
    // //Calcula C=A*B
    // for (i = 0; i < tam; i++)
    //     for (j = 0; j < tam; j++)
    //         for (k = 0; k < tam; k++)
    //             hostC[i * tam + j] += hostA[i * tam + k] * hostB[k * tam + j];
    // --------------------------------------------------

    // gpuErrchk(cudaDeviceSynchronize()); // Desnecessário, porque o cudaMemcpy já tem uma barreira implicita
    gpuErrchk(hipMemcpy(hostC, devC, tam * tam * sizeof(double), hipMemcpyDeviceToHost));

    // Imprime o resultado
    debugCode({
        printf("Matriz C:\n");
    });
    for (i = 0; i < tam; i++)
    {
        for (j = 0; j < tam; j++)
            printf("%.1lf ", hostC[i * tam + j]);
        printf("\n");
    }

    // Desaloca as matrizes do device
    gpuErrchk(hipFree(devA));
    gpuErrchk(hipFree(devB));
    gpuErrchk(hipFree(devC));

    // Desaloca as matrizes do host
    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
