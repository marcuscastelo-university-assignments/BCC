#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG_ACTIVE 0
#define debugCode(code)   \
    do                    \
    {                     \
        if (DEBUG_ACTIVE) \
        {                 \
            code;         \
        }                 \
    } while (0)

#define gpuErrchk(ans)                        \
    do                                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    } while (0)

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void multiplica_matriz_passo1(double *devA, double *devB, double *devHC, size_t tam)
{
    size_t i, j, k;
    i = blockIdx.y;
    j = blockIdx.x;
    k = threadIdx.x;

    if (k != 1)
        return;

    devHC[k * (tam * tam) + i * tam + j] = devA[i * tam + k] * devB[k * tam + j];
    printf("C[%llu][%llu] = A[%llu][%llu] * B[%llu][%llu] = %f\n", i, j, i, k, k, j, devHC[k * (tam * tam) + i * tam + j]);
    __syncthreads();
}

__global__ void multiplica_matriz_passo2(double *devA, double *devB, double *devHC, size_t tam)
{
    size_t i = blockIdx.y;
    size_t j = blockIdx.x;
    size_t k = threadIdx.x;

    int activeThreads = blockDim.x;
    size_t step = 1;

    size_t matrixSize = tam * tam;

#define hc(i, j, k) devHC[(i)*tam + (j) + (k)*matrixSize]

    while (k < activeThreads)
    {
        size_t first = k * step * 2;
        size_t second = first + step;
        if (second >= tam || first >= tam)
            break;

        hc(i, j, first) += hc(i, j, second);

        step *= 2;
        activeThreads /= 2;

        __syncthreads();
    }

#undef hc
}

__global__ void checkMultOk(double *devA, double *devB, double *devHC, size_t tam)
{
    int i,j,k;
    for (k = 0; k < tam; k++)
    {
        for (i = 0; i < tam; i++)
        {
            for (j = 0; j < tam; j++)
            {
                if (devHC[k * tam * tam + i * tam + j] != devA[i * tam + k] * devB[k * tam + j])
                {
                    printf("Erro na multiplicação de matrizes!\n");
                }
            }
        }
    }
}

int main(int argc, char **argv)
{
    // Declara as matrizes
    double *hostA, *hostB, *hostC;
    double *devA, *devB, *devHC;
    // Declara as variáveis de tamanho e índice
    size_t tam, i, j, k;

    // Lê a dimensão da matriz
    fscanf(stdin, "%d\n", &tam);

    // Aloca as matrizes no host
    hostA = (double *)calloc(tam * tam, sizeof(double));
    hostB = (double *)calloc(tam * tam, sizeof(double));
    hostC = (double *)calloc(tam * tam, sizeof(double));

    // Aloca as matrizes no device
    gpuErrchk(hipMalloc((void **)&devA, tam * tam * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&devB, tam * tam * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&devHC, tam * tam * tam * sizeof(double)));

    // Lê as matrizes A e B do stdin
    for (i = 0; i < tam; i++)
        for (j = 0; j < tam; j++)
            fscanf(stdin, "%lf ", &hostA[i * tam + j]);
    for (i = 0; i < tam; i++)
        for (j = 0; j < tam; j++)
            fscanf(stdin, "%lf ", &hostB[i * tam + j]);

    debugCode({
        printf("Matriz A:\n");
        for (i = 0; i < tam; i++)
        {
            for (j = 0; j < tam; j++)
                printf("%.1lf ", hostA[i * tam + j]);
            printf("\n");
        }

        printf("Matriz B:\n");
        for (i = 0; i < tam; i++)
        {
            for (j = 0; j < tam; j++)
                printf("%.1lf ", hostB[i * tam + j]);
            printf("\n");
        }
    });

    // Copia as matrizes para o device
    gpuErrchk(hipMemcpy(devA, hostA, tam * tam * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devB, hostB, tam * tam * sizeof(double), hipMemcpyHostToDevice));

    dim3 blocks(tam, tam, 1);
    int threads = tam;

    // Executa o código CUDA (kernel multiplica_matriz)
    multiplica_matriz_passo1<<<blocks, threads>>>(devA, devB, devHC, tam);
    gpuErrchk(hipDeviceSynchronize());
    checkMultOk<<<1,1>>>(devA, devB, devHC, tam);

    

    // multiplica_matriz_passo2<<<blocks, (threads+1)/2>>>(devA, devB, devHC, tam);

    // --------------------------------------------------
    // Código antigo (sequencial)
    // //Calcula C=A*B
    // for (i = 0; i < tam; i++)
    //     for (j = 0; j < tam; j++)
    //         for (k = 0; k < tam; k++)
    //             hostC[i * tam + j] += hostA[i * tam + k] * hostB[k * tam + j];
    // --------------------------------------------------

    // gpuErrchk(cudaDeviceSynchronize()); // Desnecessário, porque o cudaMemcpy já tem uma barreira implicita
    gpuErrchk(hipMemcpy(hostC, devHC, tam * tam * sizeof(double), hipMemcpyDeviceToHost));

    // Imprime o resultado
    debugCode({
        printf("Matriz C:\n");
    });
    for (i = 0; i < tam; i++)
    {
        for (j = 0; j < tam; j++)
            printf("%.1lf ", hostC[i * tam + j]);
        printf("\n");
    }

    // Desaloca as matrizes do device
    gpuErrchk(hipFree(devA));
    gpuErrchk(hipFree(devB));
    gpuErrchk(hipFree(devHC));

    // Desaloca as matrizes do host
    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
