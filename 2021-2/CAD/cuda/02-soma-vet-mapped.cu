/*
Faz a soma dos elementos de dois vetores 

Exemplifica o uso de memoria mapeada com cudaHostAlloc() usando
o parametro cudaHostAllocMapped para alocar memoria 
tanto no host quanto no device. Copias entre host e device sao
implicitas, igual aa memoria unificada.
cudaDeviceSynchronize() antes da impressao do resultado se faz
necessaria, caso contrário o resultado deve sair errado.

Para compilar: nvcc 02-soma-vet-mapped.cu -o 02-soma-vet-mapped
Para executar: ./02-soma-vet-mapped

OBS: os valores de tamanho do vetor e o conteudo do vetor 
     estao fixos no codigo
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void soma(int *vetorA, int *vetorB,int *vetorC,int tam)
{ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < tam) 
    {
        vetorC[idx]=vetorA[idx]+vetorB[idx];
    }
}

int main(int argc,char **argv)
{
    int i,*vetorA,*vetorB,*vetorC,threadsPerBlock,blocksPerGrid; 

    int tam = 16; //5000;
    //Define a quantidade de threads por bloco
    threadsPerBlock = 256; 

    //Aloca os vetores no host e no device (memória mapeada em endereço virtual unificado)
    hipHostAlloc((void**)&vetorA,tam*(sizeof(int)),hipHostMallocMapped); 
    hipHostAlloc((void**)&vetorB,tam*(sizeof(int)),hipHostMallocMapped); 
    hipHostAlloc((void**)&vetorC,tam*(sizeof(int)),hipHostMallocMapped); 
    
    //Preenche os vetores no host
    for(i=0;i<tam;i++)
	{
        vetorA[i]=i;
        vetorB[i]=0;  //-i;
    }
    
    //Define a quantidade de blocos por grade
    blocksPerGrid=(tam+threadsPerBlock-1)/threadsPerBlock; 

    //Invoca o kernel com blocksPerGrid blocos e threadsPerBlock threads
    
    soma <<<blocksPerGrid,threadsPerBlock>>> (vetorA,vetorB,vetorC,tam); 

	hipDeviceSynchronize();
	
    //Imprime o resultado no host
    for(i=0;i<tam;i++)
	{
       printf("%d ",vetorC[i]);
    }
	printf("\n");
	

    //Desaloca os vetores no host e no device
    hipHostFree(vetorA);
    hipHostFree(vetorB);
    hipHostFree(vetorC);
}